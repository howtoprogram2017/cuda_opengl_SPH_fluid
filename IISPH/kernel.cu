#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "timer.h"
#include <GL/glew.h>
#include <stdio.h>
extern struct hipGraphicsResource *cuda_vbo_resource;
extern "C" hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
extern "C" hipError_t subWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void subKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void advectParticles_k(float *p, float * x) {
	int i = threadIdx.x;
	float po = p[i];
	if (po > 0.5f)
		po = -0.5f;
	p[i] = po+0.01;
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	Timer timer;
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;
	timer.start();
	cudaStatus = hipSetDevice(0);
	timer.stop();
	printf("time: %d ms\n", timer.duration());
	// Choose which GPU to run on, change this on a multi-GPU system.
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	timer.start();

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	timer.stop();
	printf("time: %d ms\n", timer.duration());
	

	// Launch a kernel on the GPU with one thread for each element
	
	timer.start();
		addKernel << <1, size >> >(dev_c, dev_a, dev_b);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	timer.stop();
	printf("time: %d ms\n", timer.duration());

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

hipError_t subWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	// Choose which GPU to run on, change this on a multi-GPU system.
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	hipEvent_t start, stop;
	float gpu_time = 0.0f;
	(hipEventCreate(&start));
	(hipEventCreate(&stop));
	hipEventRecord(start, 0);
	subKernel << <1, size >> >(dev_c, dev_a, dev_b);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	(hipEventElapsedTime(&gpu_time, start, stop));
	printf("Time spent: %.5f\n", gpu_time);
	// Launch a kernel on the GPU with one thread for each element.

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
extern "C"
void advectParticles(GLuint vbo, float *v)
{
	//dim3 grid((dx / TILEX) + (!(dx%TILEX) ? 0 : 1), (dy / TILEY) + (!(dy%TILEY) ? 0 : 1));
	//dim3 tids(TIDSX, TIDSY);

	float *p;  //gl data
	hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	//getLastCudaError("hipGraphicsMapResources failed");

	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&p, &num_bytes,
		cuda_vbo_resource);
	//getLastCudaError("hipGraphicsResourceGetMappedPointer failed");

	advectParticles_k << <1,9 >> >(p, v);
	//getLastCudaError("advectParticles_k failed.");

	hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
	//getLastCudaError("hipGraphicsUnmapResources failed");
}