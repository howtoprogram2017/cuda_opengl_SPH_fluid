#include "hip/hip_runtime.h"



#include "fluid_system.cuh"
//#include "device_atomic_functions.hpp"
#include <vector>
#include <thrust\reduce.h>
#include <thrust\functional.h>
#include <thrust\execution_policy.h>


#include <iostream>
using namespace std;
extern vector<double3> boundaryParticles;


bufflist fbuf;
uint Location[2];
uint ParticleVAO[2];
//#ifndef __HIPCC__
//#define __HIPCC__

#define UNDEF_GRID -1
#define ITERATION_MAX_NUM 20000000
extern "C" hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
extern struct hipGraphicsResource *cuda_vbo_resource[2];
 __constant__ ParticleParams _param;
 //struct ParticleParams _param;
//__device__ __constant__ float deviceArray[10];

 const float radius = 0.025;
 const float smoothRadius = radius * 4;
 const float densityRatio = 1;   //control neighborNum
 const float GridSize = smoothRadius / densityRatio;
 const float3 minGridCorner = { -0.5,-0.5,-0.5 };
 const float3 maxGridCorner = { 0.5,0.5,0.5 };
 const float3 OuterGridRange = maxGridCorner - minGridCorner + (2.0 * smoothRadius)*make_float3(1.0, 1.0, 1.0); //FLOAT3_ADD( FLOAT3_SUB(maxGridCorner,minGridCorner),make_float3(smoothRadius*2, smoothRadius*2, smoothRadius*2));
 const float3 minOuterBound = { minGridCorner.x - smoothRadius,minGridCorner.y - smoothRadius,minGridCorner.z - smoothRadius };
 const float3 minWaterCorner = { -0,-0.5,-0.5 };
 const float3 maxWaterCorner = { 0.5,0.5,0.5 };
 const float3 waterRange = maxWaterCorner - minWaterCorner;
 const float restDensity = 1000;
 const uint influcedParticleNum = 50;
 const float mass = 4 * M_PI / (3 * influcedParticleNum)*pow(smoothRadius, 3.0f)*restDensity;
 float initialDistance = pow(mass / restDensity, 1.0 / 3.0);

 //initialized in setup function
 int3 outerGridDim;
 uint particleNum;
 int outerGridNum; //including ghost particles


 int ghostnum;
dim3 blocksize_p((uint)ceil(waterRange.x / initialDistance), (uint)ceil(waterRange.y / initialDistance));
dim3 gridsize_p((uint)ceil(waterRange.z / initialDistance)); //1 dimension

//dim3 blocksize_grid(outerGridDim.x, outerGridDim.y);
dim3 gridsize_grid(outerGridDim.z);

#define ErrorBound 0.02

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	Timer timer;
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;
	timer.start();
	cudaStatus = hipSetDevice(0);
	timer.stop();
	//printf("time: %d ms\n", timer.duration());
	// Choose which GPU to run on, change this on a multi-GPU system.
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	timer.start();

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	timer.stop();
	printf("time: %d ms\n", timer.duration());
	

	// Launch a kernel on the GPU with one thread for each element
	
	timer.start();
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	timer.stop();
	printf("time: %d ms\n", timer.duration());

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

inline __device__ __host__ int getGrid(float3& pos) {
	int gridx = (pos.x - _param.minOuterBound.x) / _param._GridSize;
	int gridy = (pos.y - _param.minOuterBound.y) / _param._GridSize;
	int gridz = ((pos.z - _param.minOuterBound.z) / _param._GridSize);
	if (gridx >= 0 && gridx < _param.outerGridDim.x&&gridy >= 0 && gridy < _param.outerGridDim.y&&gridz >= 0 && gridz < _param.outerGridDim.z)
		return  gridz+gridy*_param.outerGridDim.z+gridx*_param.outerGridDim.z*_param.outerGridDim.y;
	else
		return UNDEF_GRID;

}
__device__ inline void boxBoundaryForce(const float3& position, float3& force)
{
	const float  sim_scale = 1;
	const float3 vec_bound_min = _param._minGridCorner;
	const float3 vec_bound_max = _param._maxGridCorner ;
	float  param_force_distance = 0.015;
	float param_max_boundary_force = 2.0;
	float param_inv_force_distance = 1.0/param_force_distance;
	if (position.x < vec_bound_min.x + param_force_distance)
	{
	float3 boundForce= (FLOAT3_MUL_SCALAR( make_float3(1.0, 0.0, 0.0) , ((vec_bound_min.x + param_force_distance - position.x) * param_inv_force_distance * 2.0 * param_max_boundary_force)));
	force = FLOAT3_ADD(force,boundForce);
	}
	if (position.x > vec_bound_max.x - param_force_distance)
	{
		float3 boundForce = FLOAT3_MUL_SCALAR(make_float3(-1.0, 0.0, 0.0) ,((position.x + param_force_distance - vec_bound_max.x) * param_inv_force_distance * 2.0 * param_max_boundary_force));
		force = FLOAT3_ADD(force, boundForce);
	}

	if (position.y < vec_bound_min.y + param_force_distance)
	{
		float3 boundForce = FLOAT3_MUL_SCALAR(make_float3(0.0, 1.0, 0.0) , ((vec_bound_min.y + param_force_distance - position.y) * param_inv_force_distance * 2.0 * param_max_boundary_force));
		force = FLOAT3_ADD(force, boundForce);
	}
	if (position.y > vec_bound_max.y - param_force_distance)
	{
		float3 boundForce = FLOAT3_MUL_SCALAR(make_float3(0.0, -1.0, 0.0) , ((position.y + param_force_distance - vec_bound_max.y) * param_inv_force_distance * 2.0 * param_max_boundary_force));
		force = FLOAT3_ADD(force, boundForce);
	}

	if (position.z < vec_bound_min.z + param_force_distance)
	{
		float3 boundForce = FLOAT3_MUL_SCALAR(make_float3(0.0, 0.0, 1.0) , ((vec_bound_min.z + param_force_distance - position.z) * param_inv_force_distance * 2.0 * param_max_boundary_force));
		force = FLOAT3_ADD(force, boundForce);
	}

	if (position.z > vec_bound_max.z - param_force_distance)
	{
		float3 boundForce = FLOAT3_MUL_SCALAR(make_float3(0.0, 0.0, -1.0) , ((position.z + param_force_distance - vec_bound_max.z) * param_inv_force_distance * 2.0 * param_max_boundary_force));
		force = FLOAT3_ADD(force, boundForce);
	}
}
int getGridCpu(float3& pos) {
	int gridx = (pos.x - minOuterBound.x) / GridSize;
	int gridy = (pos.y - minOuterBound.y) / GridSize;
	int gridz = ((pos.z - minOuterBound.z) / GridSize);
	if (gridx >= 0 && gridx < outerGridDim.x&&gridy >= 0 && gridy < outerGridDim.y&&gridz >= 0 && gridz < outerGridDim.z)
		return  gridz + gridy * outerGridDim.z + gridx * outerGridDim.z*outerGridDim.y;
	else
		return UNDEF_GRID;

}
float poly6kernelGradientCpu(float dist) {
	if (dist > _param.smooth_radius)
		return 0;
	float ratio = dist / _param.smooth_radius;
	float tmp = 1 - ratio * ratio;
	return _param.poly6kernelGradient*(-ratio * tmp*tmp);
}
inline __host__ __device__ float poly6kernelVal(float dist) {
	if (dist > _param.smooth_radius)
		return 0;
	float ratio = dist / _param.smooth_radius;
	float tmp = 1 - ratio * ratio;
	return _param.poly6kernel*(tmp*tmp*tmp);
}
inline __host__ __device__ float poly6kernelGradient(float dist) {
	if (dist > _param.smooth_radius)
		return 0;
	float ratio = dist / _param.smooth_radius;
	float tmp = 1 - ratio * ratio;
	return _param.poly6kernelGradient*(-ratio * tmp*tmp);
}
inline __host__ __device__ float spikykernelGradient(float dist) {
	if (dist > _param.smooth_radius)
		return 0;
	float ratio = dist / _param.smooth_radius;
	float tmp = 1 - ratio * ratio;
	return _param.spikykernelGradient*(-1.0 * tmp*tmp);
}

__global__ void CountParticleInGrid(float3* p,bufflist fbuf) {
	//int i = threadIdx.z * 25 + threadIdx.y * 5 + threadIdx.x;
	int i =  blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
	float3 point = p[i];
	const float3 vec_bound_min = _param._minGridCorner;
	const float3 vec_bound_max = _param._maxGridCorner;
	//if (point.x<vec_bound_min.x || point.x>vec_bound_max.x ||
	//	point.y<vec_bound_min.y || point.y>vec_bound_max.y ||
	//	point.z<vec_bound_min.z || point.x>vec_bound_max.z) {
	//	//atomicAdd(&fbuf.max_predicted_density[0], 1);
	//}
	//else
	{
		int gridIndex = getGrid(point);
		if (gridIndex == UNDEF_GRID)
			printf("error grid\n");
		fbuf.particle_grid_cell_index[i] = gridIndex;
		fbuf.grid_particle_offset[i] = atomicAdd(&fbuf.grid_particles_num[gridIndex], 1);
	}
	

}

void ComputeDensityErrorFactor(vector<float3>& pos, int i) {
	float GradWDot = 0;
	float3 GradW = make_float3(0,0,0);
	for (int j = 0; j < pos.size(); j++) {
		if (i == j)
			continue;
		float3 pos_i_minus_j = (pos[i]-pos[j]);
		float dist_square = dot(pos_i_minus_j,pos_i_minus_j);
		float dist = sqrtf(dist_square);
		float3 gradVec = (pos_i_minus_j* poly6kernelGradientCpu(dist)/dist);
		GradWDot += dot(gradVec, gradVec);
		GradW = (GradW+ gradVec);
	}
	
	float factor = _param.mass * _param.mass * _param.time_step * _param.time_step/(_param.rest_density*_param.rest_density);
	float gradWTerm = -dot(GradW,GradW) - GradWDot;
	_param.param_density_error_factor = -1.0 / (factor*gradWTerm);
}

//
void addPointFormTriangle(float3 p1,float3 p2,float3 p3,int lod,vector<float3>&pointSet) {
	for (int i = 0; i <= lod; i++) {
		for (int j = 0; i + j <= lod; j++) {
			pointSet.push_back(p1*(float(i)/float(lod))+p2* (float(j) / float(lod)) +p3* (float(lod-i-j) / float(lod)));
		}
	}

}
void Setup() {
	//CUDA_SAFE_CALL(hipMalloc((void**)&testdata, 9*sizeof(float)));
	//memset(testdata, 0, 9 * sizeof(float));
	outerGridDim =  { (int)ceil(OuterGridRange.x / GridSize),(int)ceil(OuterGridRange.y / GridSize) ,(int)ceil(OuterGridRange.z / GridSize) };
	 particleNum = (int)ceil(waterRange.x / initialDistance)*(int)ceil(waterRange.y / initialDistance)*(int)ceil(waterRange.z / initialDistance);  //a particle per grid

	 outerGridNum = outerGridDim.x*outerGridDim.y*outerGridDim.z; //including ghost particles
	  blocksize_p=dim3((uint)ceil(waterRange.x / initialDistance), (uint)ceil(waterRange.y / initialDistance));
	  gridsize_p=dim3((uint)ceil(waterRange.z / initialDistance)); //1 dimension

																 //dim3 blocksize_grid(outerGridDim.x, outerGridDim.y);
	  gridsize_grid=dim3((uint)outerGridDim.z);
	glGenBuffers(2, &Location[0]);
	glGenVertexArrays(2, &ParticleVAO[0]);
	vector<float3> initialPos = vector<float3>();
	vector<float3> ghostPos = vector<float3>();
	int total = particleNum;
	int numX = (int)ceil(waterRange.x / initialDistance);
	int numY = (int)ceil(waterRange.y / initialDistance);
	int numZ = (int)ceil(waterRange.z / initialDistance);
	float offsetX = (waterRange.x-(numX-1)*initialDistance) / 2.0f;
	float offsetY = (waterRange.y - (numY - 1)*initialDistance) / 2.0f;
	float offsetZ = (waterRange.z - (numZ - 1)*initialDistance) / 2.0f;
	float3 startPos = make_float3(offsetX+minWaterCorner.x,offsetY+minWaterCorner.y,offsetZ+minWaterCorner.z);
	for(int i=0;i<numX;i++)
		for (int j = 0; j<numY; j++)
			for (int k = 0; k < numZ; k++) {
				int index = k + numZ * j + i * numZ*numY;
				float3 move = {i*initialDistance,j*initialDistance,k*initialDistance};
				float3 pos = (startPos+move);
				
				initialPos.push_back(pos);
			}
	float posx = startPos.x;
	float posy = startPos.y;
	float posz = startPos.z;
	posx = posx - initialDistance*(int)((posx - minGridCorner.x) / initialDistance);
	posy = posz - initialDistance * (int)((posy - minGridCorner.y) / initialDistance);
	posz = posz - initialDistance * (int)((posz - minGridCorner.z) / initialDistance);

	while (true)
	{
		if (posx - initialDistance > minOuterBound.x)
			posx -= initialDistance;
		else break;
	}
	while (true)
	{
		if (posy - initialDistance > minOuterBound.y)
			posy -= initialDistance;
		else break;
	}
	while (true)
	{
		if (posz - initialDistance > minOuterBound.z)
			posz -= initialDistance;
		else break;
	}
	float vertices[] = {
		-0.52f, -0.52f, -0.52f,  
		0.52f,  0.52f, -0.52f,  
		0.52f, -0.52f, -0.52f, 
		-0.52f, -0.52f, -0.52f,  
		-0.52f,  0.52f, -0.52f,  
		0.52f,  0.52f, -0.52f,  

		-0.52f, -0.52f,  0.52f, 
		0.52f, -0.52f,  0.52f,  
		0.52f,  0.52f,  0.52f,  
		0.52f,  0.52f,  0.52f,  
		-0.52f,  0.52f,  0.52f,  
		-0.52f, -0.52f,  0.52f, 

		-0.52f,  0.52f, -0.52f,  
		-0.52f, -0.52f, -0.52f,  
		-0.52f,  0.52f,  0.52f,  
		-0.52f, -0.52f,  0.52f,  
		-0.52f,  0.52f,  0.52f,  
		-0.52f, -0.52f, -0.52f,  

		0.52f,  0.52f, -0.52f,  
		0.52f,  0.52f,  0.52f,  
		0.52f, -0.52f, -0.52f,  
		0.52f, -0.52f,  0.52f, 
		0.52f, -0.52f, -0.52f,  
		0.52f,  0.52f,  0.52f,   
		//bottom
		-0.52f, -0.52f, -0.52f,   
		0.52f, -0.52f, -0.52f,   
		0.52f, -0.52f,  0.52f,   
		-0.52f, -0.52f,  0.52f,   
		-0.52f, -0.52f, -0.52f,   
		0.52f, -0.52f,  0.52f,
		-0.52f, 0.52f, -0.52f,
		0.52f, 0.52f, -0.52f,
		0.52f, 0.52f,  0.52f,
		-0.52f, 0.52f,  0.52f,
		-0.52f, 0.52f, -0.52f,
		0.52f, 0.52f,  0.52f,
	};
	//for (int i = 0; i < 6; i++) {
	//	float dir1 = startBound[i].x == startBound[i].y;

	//}

	/*for (float px=posx; px < maxGridCorner.x + smoothRadius; px += initialDistance) {
		for (float py=posy; py < maxGridCorner.y + smoothRadius; py += initialDistance) {
			for (float pz=posz ; pz < maxGridCorner.z + smoothRadius; pz += initialDistance) {
				if (px > minGridCorner.x  && px<maxGridCorner.x  &&
					py>minGridCorner.y  && py<maxGridCorner.x  &&
					pz>minGridCorner.z && pz < maxGridCorner.x)
					continue;

				ghostPos.push_back({ px,py,pz });
			}
		}
	}*/
	ghostPos.resize(boundaryParticles.size());
	for (int i = 0; i < boundaryParticles.size(); i++)
		ghostPos[i] = make_float3((float)boundaryParticles[i].x, (float)boundaryParticles[i].y, (float)boundaryParticles[i].z);
	/*int lod = (int)ceilf(1.02/initialDistance);
	for (int i = 0; i < 12;i++ ) {
		addPointFormTriangle({ vertices[9 * i],vertices[9 * i + 1],vertices[9 * i + 2] },
			{ vertices[9 * i + 3],vertices[9 * i + 4],vertices[9 * i + 5] },
			{ vertices[9 * i + 6 ],vertices[9 * i + 7],vertices[9 * i + 8] }, lod,ghostPos);
	}*/

	//ghostPos.push_back({ 1.0,0.0,0.0 });
	for (int i = 0; i < 2; i++) {
		glBindVertexArray(ParticleVAO[i]);
		glBindBuffer(GL_ARRAY_BUFFER, Location[i]);
		glBufferData(GL_ARRAY_BUFFER, sizeof(float3)*initialPos.size(), &initialPos[0], GL_STATIC_DRAW);
		//glVertexAttribPointer(1,)
		glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, (void*)0);
		glEnableVertexAttribArray(0);
		hipGraphicsGLRegisterBuffer(&cuda_vbo_resource[i], Location[i], hipGraphicsRegisterFlagsNone);
	}
	
	hipDeviceSynchronize();
	//hipMemcpyToSymbol(HIP_SYMBOL(&_param), &cpuParam, sizeof(cpuParam), 0);
	//__constant__ ParticleParams* _param1;
	uint index = 0;
	float gravity=9.0;
	//float mass = 1.0;
	float poly6kernel = 315.0f / (64.0f*M_PI*pow(smoothRadius, 3.0f));
	float poly6kernelGrad = 945.0f / (32.0f*M_PI*pow(smoothRadius, 4.0f));
	float boudary_force_factor = 25.0;
	float time_step = 0.0004;
	float spikykernelGrad = 45.0f / (M_PI*pow(smoothRadius, 4.0f));
	
	//{ minGridCorner ,maxGridCorner,GridIndexRange,GridSize,particleNum,gravity,mass,time_step,smoothRadius,restDensity,poly6kernel,poly6kernelGrad,density_error_factor,boudary_force_factor} ;
	_param._minGridCorner = minGridCorner;
	_param._maxGridCorner = maxGridCorner;
	_param._GridSize = GridSize;
	_param.outerGridDim = outerGridDim;
	_param.particleNum = particleNum;
	_param.gravity = gravity;
	_param.mass = mass;
	_param.time_step = time_step;
	_param.smooth_radius = smoothRadius;
	_param.rest_density = restDensity;
	_param.poly6kernel = poly6kernel;
	_param.poly6kernelGradient = poly6kernelGrad;
	_param.minOuterBound =minOuterBound;
	_param.spikykernelGradient = spikykernelGrad;


	for (int i = -1; i < 2; i++)
		for (int j = -1; j < 2; j++)
			for (int k = -1; k < 2; k++)
				_param._neighbor_off[index++] = { k,j,i };
	ComputeDensityErrorFactor(initialPos,initialPos.size()/2);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(_param), &_param, sizeof(ParticleParams), 0, hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceArray), hostArray, sizeof(float)*10, 0, hipMemcpyHostToDevice));

	//memset(initialPos,0.2, total*sizeof(float3));
	
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.grid_particle_offset, particleNum * sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.particle_grid_cell_index, particleNum * sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.particle_grid_cell_index_update, particleNum * sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.vel_old, particleNum * sizeof(float3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.vel_update, particleNum * sizeof(float3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.pos_update, particleNum * sizeof(float3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.sort_index, particleNum * sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.force, particleNum * sizeof(float3)));
	CUDA_SAFE_CALL(hipMemset(fbuf.vel_old, 0, particleNum * sizeof(float3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.predicted_density, particleNum * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.correction_pressure_force, particleNum * sizeof(float3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.correction_pressure, particleNum * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.max_predicted_density, sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.densityError, particleNum * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(fbuf.densityError, 0, particleNum * sizeof(float)));
	
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.grid_off, outerGridNum * sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.grid_particles_num, outerGridNum * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(fbuf.grid_particles_num,0,outerGridNum*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.ghost_grid_off, outerGridNum* sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.ghost_grid_particles_num, outerGridNum * sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.ghost_volum, ghostPos.size() * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.ghost_pos, ghostPos.size()* sizeof(float3)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.test_buff, particleNum * sizeof(float3)));
	ghostnum = ghostPos.size();
	vector<int> ghost_particle_index_grid(ghostPos.size());
	vector<int> ghost_particle_grid_index(ghostPos.size());
	vector<int> sorted_index(ghostPos.size());
	vector<float3> pos_tmp(ghostPos.size());
	vector<int> ghost_grid_particles_num(outerGridNum,0);
	vector<int> ghost_grid_off(outerGridNum+1);
	vector<float> ghost_vol(ghostnum);
	for (int i = 0; i < ghostPos.size();i++) {
		float3 pos = ghostPos[i];
		int index = getGridCpu(pos);
		if (index != UNDEF_GRID) {
			ghost_particle_grid_index[i] = index;
			ghost_particle_index_grid[i] = ghost_grid_particles_num[index];
			ghost_grid_particles_num[index]++;
		}
	}
	int grid_off = 0;
	for (int i = 0; i < outerGridNum; i++) {
		ghost_grid_off[i] = grid_off;
		grid_off += ghost_grid_particles_num[i];
	}
	ghost_grid_off.push_back(grid_off);
	for (int i = 0; i < ghostPos.size(); i++) {
		int cell_index = ghost_particle_grid_index[i];
		sorted_index[i] = ghost_grid_off[cell_index] + ghost_particle_index_grid[i];
	}
	for (int i = 0; i < ghostPos.size(); i++) {
		int index = sorted_index[i];
		pos_tmp[index] = ghostPos[i];
	}
	for (int i = 0; i < ghostPos.size(); i++) {
		int i_cell_index = getGridCpu(pos_tmp[i]);
		int3 GridnumRange = _param.outerGridDim;
		int cell_z = i_cell_index % (GridnumRange.z);
		i_cell_index /= GridnumRange.z;
		int cell_y = i_cell_index % (GridnumRange.y);
		int cell_x = i_cell_index / GridnumRange.y;
		//int index = getGridCpu(ghostPos[i]);
		float Wsum = 0.0;
		for (int cell = 0; cell < neighborGridNum; cell++)
		{
			int cell_neighbor_x = cell_x + _param._neighbor_off[cell].x;
			int cell_neighbor_y = cell_y + _param._neighbor_off[cell].y;
			int cell_neighbor_z = cell_z + _param._neighbor_off[cell].z;
			if (cell_neighbor_x < 0 || cell_neighbor_x >= _param.outerGridDim.x || cell_neighbor_y < 0 || cell_neighbor_y >= _param.outerGridDim.y || cell_neighbor_z < 0 || cell_neighbor_z >= _param.outerGridDim.z)
				continue;
			int neighbor_cell_index = cell_neighbor_z + cell_neighbor_y * GridnumRange.z + cell_neighbor_x * GridnumRange.z*GridnumRange.y;
			int ghost_cell_start = ghost_grid_off[neighbor_cell_index];
			int ghost_cell_end = ghost_grid_off[neighbor_cell_index+1];
			for (int cndx = ghost_cell_start; cndx < ghost_cell_end; cndx++)
			{
				//force.y++;
				int j = cndx;
				float3 vector_i_minus_j = (pos_tmp[i]- pos_tmp[j]);
				const float jdist = length(vector_i_minus_j);
				if (jdist < _param.smooth_radius)
				{
					//float jdist = sqrt(dist_square);
					float kernel = poly6kernelVal(jdist);
					Wsum += kernel*_param.mass/_param.rest_density;

				}
			}

		}
		ghost_vol[i] = 1.0/ Wsum;
	}
	CUDA_SAFE_CALL(hipMemcpy(fbuf.ghost_volum, &ghost_vol[0], sizeof(float)*ghostnum, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fbuf.ghost_grid_off,&ghost_grid_off[0],sizeof(int)*outerGridNum,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fbuf.ghost_grid_particles_num, &ghost_grid_particles_num[0], sizeof(int)*outerGridNum, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(fbuf.ghost_pos, &pos_tmp[0], sizeof(float3)*ghostPos.size(), hipMemcpyHostToDevice));
	ghostnum = ghostPos.size();

}

void ClearSystem() {

}
void computeCUDAGridBlockSize(int numParticles, int blockSize, int &numBlocks, int &numThreads);
extern "C"
void ParticleSetupCUDA(int num, int gsrch, int3 res, float3 size, float3 delta, float3 gmin, float3 gmax, int total, int chk, float grid_cell_size, float param_kernel_self)
{
	

	//deallocBlockSumsInt();
	//preallocBlockSumsInt(fcudaParams.param_grid_total);
}
//helper function
int iDivUp(int a, int b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}
void computeCUDAGridBlockSize(int numParticles, int blockSize, int &numBlocks, int &numThreads)
{
	numThreads = min(blockSize, numParticles);
	numBlocks = iDivUp(numParticles, numThreads);
}
uint getLocation() {
	return ParticleVAO[0];
}
void swapBuff() {
	swap(cuda_vbo_resource[0], cuda_vbo_resource[1]);
	swap(Location[0], Location[1]);
	swap(ParticleVAO[0], ParticleVAO[1]);
	swap(fbuf.vel_old, fbuf.vel_update);
	//swap(fbuf.particle_grid_cell_index, fbuf.particle_grid_cell_index_update);
}
void CountParticles(float3* input) {
	//dim3 threadperBlock(numRangeX,numRangeY ,numRangeZ);
	CUDA_SAFE_CALL(hipMemset(fbuf.grid_particles_num, 0, outerGridNum*sizeof(uint) ));
	//hipMemset(fbuf.max_predicted_density, 0, sizeof(float));

	CountParticleInGrid <<<gridsize_p, blocksize_p >> > (input, fbuf);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	auto input2 = vector<int>(outerGridNum);
	float* input1 = new float(0);
#ifdef TEST
	hipMemcpy(&input1[0], fbuf.max_predicted_density, sizeof(int), hipMemcpyDeviceToHost);
	cout << "error" << *input1 << endl;
	hipMemcpy(&input2[0], fbuf.grid_particles_num, outerGridNum * sizeof(int), hipMemcpyDeviceToHost);

	for (auto a : input2)
		cout << a << " ";
	cout << endl;
#endif // TEST

	
	hipDeviceSynchronize();
}

__global__ void rearrange(bufflist fbuf,float3* pos_old) {
	int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
	uint index = fbuf.sort_index[tid];
	fbuf.vel_update[index] = fbuf.vel_old[tid];
	fbuf.pos_update[index] = pos_old[tid];
	fbuf.particle_grid_cell_index_update[index] = fbuf.particle_grid_cell_index[tid];
}
__global__ void sortIndex(bufflist fbuf) {
	int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
	uint cell_index = fbuf.particle_grid_cell_index[tid];
	uint particle_index = fbuf.grid_off[cell_index] + fbuf.grid_particle_offset[tid];
	fbuf.sort_index[tid] = particle_index;
}
__global__ void computeOtherForce(bufflist fbuf) {
	int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
	fbuf.force[tid] = { 0,-_param.gravity,0 };
	//boxBoundaryForce(fbuf.pos_update[tid], fbuf.force[tid]);
}
__device__ void collisionHandling(float3* pos,float3* vel) {
	//const float3 vec_bound_min = _param._minGridCorner;
	//const float3 vec_bound_max = _param._maxGridCorner;
	//float damping = 0.9;

	//float reflect = 1.1;
	//	if (pos->x < vec_bound_min.x)
	//	{
	//		pos->x = vec_bound_min.x;
	//		if (vel) {
	//			float3 axis = make_float3(-1, 0, 0);
	//			*vel = FLOAT3_SUB((*vel), FLOAT3_MUL_SCALAR(axis, FLOAT3_DOT(axis, (*vel))*reflect));
	//			vel->x *= damping;
	//		}
	//	}
	//	if (pos->x > vec_bound_max.x)
	//	{
	//		pos->x = vec_bound_max.x;
	//		if (vel) {
	//			float3 axis = make_float3(1, 0, 0);
	//			*vel = FLOAT3_SUB((*vel), FLOAT3_MUL_SCALAR(axis, FLOAT3_DOT(axis, (*vel))*reflect));
	//			vel->x *= damping;
	//		}
	//	}
	//	if (pos->y < vec_bound_min.y)
	//	{
	//		pos->y = vec_bound_min.y;
	//		if (vel) {
	//			float3 axis = make_float3(0, -1, 0);
	//			*vel = FLOAT3_SUB((*vel), FLOAT3_MUL_SCALAR(axis, FLOAT3_DOT(axis, (*vel))*reflect));
	//			vel->y *= damping;
	//		}
	//	
	//	}
	//	if (pos->y > vec_bound_max.y)
	//	{
	//		pos->y = vec_bound_max.y;
	//		if (vel) {
	//			float3 axis = make_float3(0, 1, 0);
	//			*vel = FLOAT3_SUB((*vel), FLOAT3_MUL_SCALAR(axis, FLOAT3_DOT(axis, (*vel))*reflect));
	//			vel->y *= damping;
	//		}
	//		
	//	}
	//	if (pos->z < vec_bound_min.z)
	//	{
	//		pos->z = vec_bound_min.z;
	//		if (vel) {
	//			float3 axis = make_float3(0, 0, -1);
	//			*vel = FLOAT3_SUB((*vel), FLOAT3_MUL_SCALAR(axis, FLOAT3_DOT(axis, (*vel))*reflect));
	//			vel->z *= damping;
	//		}

	//	}
	//	if (pos->z > vec_bound_max.z)
	//	{
	//		pos->z = vec_bound_max.z;
	//		if (vel) {
	//			float3 axis = make_float3(0, 0, 1);
	//			*vel = FLOAT3_SUB((*vel), FLOAT3_MUL_SCALAR(axis, FLOAT3_DOT(axis, (*vel))*reflect));
	//			vel->z *= damping;
	//		}
	//	}
		/*if(vel)
		*vel = FLOAT3_MUL_SCALAR((*vel),0.99 );*/
	
}

__global__ void PredictPosition(bufflist fbuf,float3 * output_pos) {
	int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
	if (tid >= _param.particleNum)
		return;
	//const float	   sim_scale = simData.param_sim_scale;
	
	//float3 acceleration = FLOAT3_MUL_SCALAR(FLOAT3_ADD(fbuf.force[tid], fbuf.correction_pressure_force[tid]), (1.0f / _param.mass));
	float3 acceleration = (fbuf.force[tid]+ fbuf.correction_pressure_force[tid])* (1.0f / _param.mass);

	float3 predictedVelocity = (fbuf.vel_update[tid] + (acceleration* _param.time_step));

	//float3 pos = FLOAT3_ADD(fbuf.pos_update[tid] , FLOAT3_MUL_SCALAR(predictedVelocity, _param.time_step));
	float3 pos = (fbuf.pos_update[tid] + predictedVelocity * _param.time_step);
	//collisionHandling(&pos, NULL);

	output_pos[tid] = pos;
}
__global__ void ComputePredictedDensityAndPressure(bufflist fbuf,float3* predicted_pos) {
	int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
	if (tid >= _param.particleNum)
		return;

	uint i_cell_index = fbuf.particle_grid_cell_index_update[tid];
	int3 GridnumRange = _param.outerGridDim;
	int cell_z = i_cell_index % (GridnumRange.z);
	i_cell_index /= GridnumRange.z;
	int cell_y = i_cell_index % (GridnumRange.y);
	int cell_x = i_cell_index / GridnumRange.y;

	const float3 ipredicted_pos = predicted_pos[tid];
	const float  smooth_radius = _param.smooth_radius;
	const float  smooth_radius_square = smooth_radius * smooth_radius;
	//const float  sim_scale_square = simData.param_sim_scale * simData.param_sim_scale;
	const float  mass = _param.mass;
	float predictedSPHDensity = 0.0;
	for (int cell = 0; cell < neighborGridNum; cell++)
	{	
		int cell_neighbor_x = cell_x + _param._neighbor_off[cell].x;
		int cell_neighbor_y = cell_y + _param._neighbor_off[cell].y;
		int cell_neighbor_z = cell_z + _param._neighbor_off[cell].z;
		if (cell_neighbor_x < 0 || cell_neighbor_x >= _param.outerGridDim.x || cell_neighbor_y < 0 || cell_neighbor_y >= _param.outerGridDim.y || cell_neighbor_z < 0 || cell_neighbor_z >= _param.outerGridDim.z)
			continue;
		int neighbor_cell_index=cell_neighbor_z+ cell_neighbor_y* GridnumRange.z+ cell_neighbor_x* GridnumRange.z*GridnumRange.y;
		// real water particles
		if (fbuf.grid_particles_num[neighbor_cell_index] != 0)
		{
			int cell_start = fbuf.grid_off[neighbor_cell_index];
			int cell_end = cell_start + fbuf.grid_particles_num[neighbor_cell_index];

			for (int cndx = cell_start; cndx < cell_end; cndx++)
			{
				int j = cndx;
				if (tid == j)
				{
					continue;
				}
				float3 vector_i_minus_j = (ipredicted_pos- predicted_pos[j]);
				const float dx = vector_i_minus_j.x;
				const float dy = vector_i_minus_j.y;
				const float dz = vector_i_minus_j.z;
				const float dist_square_scale = dx * dx + dy * dy + dz * dz;
				if (dist_square_scale <= smooth_radius_square && dist_square_scale > 0)
				{
					//predictedSPHDensity += 1;
					const float dist = sqrt(dist_square_scale);
					float kernelValue = poly6kernelVal(dist);
					predictedSPHDensity += kernelValue * mass;
					//predictedSPHDensity += 1;
				}
			}
		}
		//ghost particles
		if (fbuf.ghost_grid_particles_num[neighbor_cell_index] > 0) {
			int cell_start = fbuf.ghost_grid_off[neighbor_cell_index];
			int cell_end = cell_start + fbuf.ghost_grid_particles_num[neighbor_cell_index];

			for (int cndx = cell_start; cndx < cell_end; cndx++)
			{
				int j = cndx;
				float3 vector_i_minus_j = FLOAT3_SUB(ipredicted_pos, fbuf.ghost_pos[j]);
				const float dist_square_scale = dot(vector_i_minus_j,vector_i_minus_j);
				if (dist_square_scale <= smooth_radius_square && dist_square_scale > 0)
				{
					//predictedSPHDensity += 1;
					/*if (tid == 0)
						printf("not correct x: %f,y: %f,z: %f\nself x:%f,y:%f,z:%f\n",
							fbuf.ghost_pos[j].x,fbuf.ghost_pos[j].y, fbuf.ghost_pos[j].z,ipredicted_pos.x,ipredicted_pos.y,ipredicted_pos.z);*/
					const float dist = sqrt(dist_square_scale);
					float kernelValue = poly6kernelVal(dist) *fbuf.ghost_volum[j] ;
					predictedSPHDensity += kernelValue * mass;
					//predictedSPHDensity = 10010.0;
					//predictedSPHDensity += 1;
				}
			}
		}

		
	}

	predictedSPHDensity += _param.poly6kernel* mass;

	 float densityError= MAX(predictedSPHDensity-_param.rest_density,0.0 );
	 fbuf.densityError[tid] = densityError;
	// fbuf.test_buff[tid].x = predictedSPHDensity;
	fbuf.correction_pressure[tid] += densityError*_param.param_density_error_factor;

	fbuf.predicted_density[tid] = predictedSPHDensity;
	//get max Error;
	//atomicMax((double*)fbuf.max_predicted_density,(double)densityError);
}
__global__ void ComputePressureForce(bufflist fbuf,float3* predicted_pos) {
		int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
		if (tid >= _param.particleNum)
			return;

		uint i_cell_index = fbuf.particle_grid_cell_index_update[tid];
		int3 GridnumRange = _param.outerGridDim;
		int cell_z = i_cell_index % (GridnumRange.z);
		i_cell_index /= GridnumRange.z;
		int cell_y = i_cell_index % (GridnumRange.y);
		int cell_x = i_cell_index / GridnumRange.y;
		if (i_cell_index == UNDEF_GRID)
			return;

		const float3 ipos = fbuf.pos_update[tid];
		const float  ipress = fbuf.correction_pressure[tid];
		const float  mass = _param.mass;
		const float  smooth_radius = _param.smooth_radius;
		const float  smooth_radius_square = smooth_radius * smooth_radius;
		const float  rest_volume = mass / _param.rest_density;
		float3 force = make_float3(0, 0, 0);
		float3 forceB = make_float3(0, 0, 0);
		for (int cell = 0; cell < neighborGridNum; cell++)
		{
			int cell_neighbor_x = cell_x + _param._neighbor_off[cell].x;
			int cell_neighbor_y = cell_y + _param._neighbor_off[cell].y;
			int cell_neighbor_z = cell_z + _param._neighbor_off[cell].z;
			if (cell_neighbor_x < 0 || cell_neighbor_x >= _param.outerGridDim.x || cell_neighbor_y < 0 || cell_neighbor_y >= _param.outerGridDim.y || cell_neighbor_z < 0 || cell_neighbor_z >= _param.outerGridDim.z)
				continue;
			int neighbor_cell_index = cell_neighbor_z + cell_neighbor_y * GridnumRange.z + cell_neighbor_x * GridnumRange.z*GridnumRange.y;
			//water particles

			int cell_start = fbuf.grid_off[neighbor_cell_index];
			int cell_end = cell_start + fbuf.grid_particles_num[neighbor_cell_index];
			for (int cndx = cell_start; cndx < cell_end; cndx++)
			{
				//force.y++;
				int j = cndx;
				if (tid == j)
				{
					continue;
				}
				float3 vector_i_minus_j = (ipos - fbuf.pos_update[j]);

				const float dist_square = dot(vector_i_minus_j, vector_i_minus_j); 
				if (dist_square < smooth_radius_square && dist_square > 0)
				{
					float jdist = sqrt(dist_square);
					float kernelGradientValue = poly6kernelGradient(jdist);
					float3 kernelGradient = ( vector_i_minus_j * kernelGradientValue/jdist);
					float grad = 0.5f * (ipress + fbuf.correction_pressure[j]) * rest_volume * rest_volume;
					force -= kernelGradient * grad;
				}
			}
			int ghost_cell_start = fbuf.ghost_grid_off[neighbor_cell_index];
			int ghost_cell_end = ghost_cell_start + fbuf.ghost_grid_particles_num[neighbor_cell_index];
			for (int cndx = ghost_cell_start; cndx < ghost_cell_end; cndx++)
			{
				////force.y++;
				//if (tid == 0)
				//	printf("dist: %f\n", cndx);
				int j = cndx;
				float3 vector_i_minus_j = ipos- fbuf.ghost_pos[j];
				const float dist = length(vector_i_minus_j);

				if (dist < smooth_radius && dist > 0)
				{
					float kernelGradientValue = poly6kernelGradient(dist);
					float3 kernelGradient = vector_i_minus_j*(( kernelGradientValue/ dist)*fbuf.ghost_volum[j]);
					float grad = 0.5f * (ipress) * rest_volume * rest_volume;
					forceB -= kernelGradient * grad;
					//force = FLOAT3_SUB(force, FLOAT3_MUL_SCALAR(kernelGradient, grad));
					
				}
			}
		}

		fbuf.correction_pressure_force[tid] = force+forceB;
		/*if (tid == 0)
			printf("pressure force fluid x: %f,y: %f,z: %f\n pressure force boudary x: %f,y: %f,z: %f\nself x:%f,y:%f,z:%f\n",
				force.x,force.y,force.z,forceB.x,forceB.y,forceB.z,ipos.x,ipos.y,ipos.z);*/
	
}
__global__ void reduceMax(float *g_idata, float *g_odata,int num) {

	extern __shared__ float sdata[];
	//// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	//unsigned int i =  threadIdx.x;
	if(tid<num)
	sdata[tid] = g_idata[tid];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = num; s > 1; s = (s+1)/2) {
		//s is odd
		if (s % 2)
		{
			if (tid < s / 2)
				sdata[tid] = MAX(sdata[tid], sdata[tid + s / 2]);
			else if (tid == s / 2)
				sdata[tid] = sdata[s-1];
		}
		else
		if (tid < s/2) {
			sdata[tid] =MAX(sdata[tid], sdata[tid + s/2]);
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[0] = sdata[0];
}
__global__ void advanceParticles(bufflist fbuf,float3* output) {
	int tid = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;
	if (tid >= _param.particleNum)
		return;

	//float3 acceleration = FLOAT3_MUL_SCALAR( FLOAT3_ADD(fbuf.force[tid] , fbuf.correction_pressure_force[tid]), 1.0/ _param.mass);
	float3 acceleration = (1.0 / _param.mass)*(fbuf.force[tid] + fbuf.correction_pressure_force[tid]);
	float3 veval = fbuf.vel_update[tid];
	veval += acceleration * _param.time_step;
	float3 pos = fbuf.pos_update[tid];
	pos += veval * _param.time_step;
	collisionHandling(&pos, &veval);

	output[tid] = pos;
	fbuf.vel_update[tid] = (veval);
}
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
float ReduceMax(float* input, float* output, int num) {
	thrust::device_ptr<float> data(input);
	float res= thrust::reduce(data, data + num
		,-1.0,
		thrust::maximum<float>()
	);
	return res;
	dim3 blocksize(ceil(sqrt(num)), ceil(sqrt(num)));
	reduceMax << <1, blocksize, num * sizeof(float) >> > (input, output,num);
	hipDeviceSynchronize();
	float max_density_error;
	hipMemcpy(&max_density_error, fbuf.max_predicted_density, sizeof(float), hipMemcpyDeviceToHost);
	////Safe
		//float res = *max_density_error;
	//delete max_density_error;
		return max_density_error;

}
void IndexSort(float3* pos_old) {
	
	prescanInt(fbuf.grid_particles_num, fbuf.grid_off,outerGridNum, outerGridDim.x, outerGridDim.y, outerGridDim.z);
	//dim3 blocksize(particleNum);

	auto input1 = vector<float3>(particleNum);
	auto input2 = vector<int>(particleNum);
	//sortIndex << <1, particleNum >> >(fbuf.grid_off, fbuf.grid_particle_offset, fbuf.particle_grid_cell_index, fbuf.sort_index);
#ifdef TEST
	hipMemcpy(&input1[0], pos_old, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
	for (auto a : input1)
		cout << a.y << " ";
	cout << "pos_old" << endl;
#endif // TEST

	
	hipDeviceSynchronize();
	sortIndex<<<gridsize_p, blocksize_p >>>(fbuf);
	//Safe
	hipDeviceSynchronize();
#ifdef TEST
	hipMemcpy(&input2[0], fbuf.sort_index, particleNum * sizeof(int), hipMemcpyDeviceToHost);
	for (auto a : input2)
		cout << a << " ";
	cout << "sort_index" << endl;
#endif // TEST

	

	//CUDA_SAFE_CALL(hipMemcpy(&input1[0], fbuf.sort_index, particleNum * sizeof(int), hipMemcpyDeviceToHost));
	//for (auto a : input1)
	//	cout << a << " ";
	rearrange<<<gridsize_p, blocksize_p >>>(fbuf,pos_old);
	

#ifdef TEST
	hipMemcpy(&input1[0], fbuf.pos_update, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
	for (auto a : input1)
		cout << a.y << " ";
	cout << "pos_update" << endl;
#endif // TEST

	
	hipDeviceSynchronize();
}
void ComputeOtherForce() {    //grivty
//	dim3 blockSize();
	computeOtherForce<<<gridsize_p,blocksize_p>>>(fbuf);
	//Safe
	auto input1 = vector<float3>(particleNum);

	//hipMemcpy(&input1[0], fbuf.force, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
	/*for (auto a : input1)
		cout << a.x << " "<<a.y<<" "<<a.z<<" ";*/
	/*for (int i = 0; i < 10; i++) {
		auto a = input1[i];
		cout << a.x << " " << a.y << " " << a.z << " ";
	}*/

	//cout << endl;
	//hipDeviceSynchronize();
}
void PredictonCorrection(float3* output) {
	bool densityErrorLarge=true;
	int cnt = 0;
	CUDA_SAFE_CALL(hipMemset(fbuf.correction_pressure,0, sizeof(float)*particleNum));
	ComputePredictedDensityAndPressure << <gridsize_p, blocksize_p >> >(fbuf, output);
	while (cnt < 1||(densityErrorLarge&&cnt<ITERATION_MAX_NUM)) {
		auto input1 = vector<float3>(particleNum);
		auto input2 = vector<float>(particleNum);

		//hipDeviceSynchronize();
#ifdef TEST
		hipMemcpy(&input1[0], fbuf.pos_update, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
		for (auto a : input1)
			cout << a.y << " ";
		cout << endl;
		hipDeviceSynchronize();
#endif // TEST

		
		
		PredictPosition<<<gridsize_p,blocksize_p>>>(fbuf,output);
#ifdef TEST
		hipMemcpy(&input1[0], output, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
		for (auto a : input1)
			cout << a.y << " ";
		cout << endl;
		hipDeviceSynchronize();
#endif // TEST

		//hipMemcpy(&input2[0], fbuf.ghost_volum, ghostnum * sizeof(float), hipMemcpyDeviceToHost);

		ComputePredictedDensityAndPressure<<<gridsize_p,blocksize_p>>>(fbuf,output);
		//Safe
		//hipMemcpy(&input2[0], fbuf.densityError, particleNum * sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(&input1[0], output, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
		////cout << "density:" << endl;
		//int ind = -1; float maxError=000.0;
		//for (int i = 0; i < particleNum; i++) {
		//	if (input2[i] > maxError) {
		//		maxError = input2[i];
		//		ind = i;
		//	}
		//}
		/*for (auto a : input2)
			cout << a << " ";
		cout << endl;*/
		//hipDeviceSynchronize();
		//CUDA_SAFE_CALL(hipMemset(fbuf.densityError, 0, sizeof(float)*particleNum));
//#ifdef TEST
			//hipMemcpy(&input1[0], fbuf.correction_pressure_force, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
		/*	cout << "correction_pressure_force" << endl;
		for (auto a : input1)
			cout << a.y << " ";
		cout << endl;
		hipDeviceSynchronize();
		hipMemcpy(&input1[0], output, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
		cout << "pos y" << endl;
		for (auto a : input1)
			cout << a.y << " ";
		cout << endl;
		hipDeviceSynchronize();*/
		
//#endif // TEST

		
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		float max_density_error;
		max_density_error = ReduceMax(fbuf.densityError, fbuf.max_predicted_density,particleNum);
		CUDA_SAFE_CALL(hipDeviceSynchronize());

		//reduceMax << <1, particleNum, particleNum * sizeof(float) >> > (fbuf.densityError, fbuf.max_predicted_density);;
		//hipMemcpy(&max_density_error, fbuf.max_predicted_density, sizeof(float), hipMemcpyDeviceToHost);
		max_density_error = MAX(0, max_density_error);
		if (max_density_error / restDensity < ErrorBound)
			densityErrorLarge = false;
		ComputePressureForce << <gridsize_p, blocksize_p >> >(fbuf,output);
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		cnt++;
	}
}
void Advance(float3* output) {
	advanceParticles << <gridsize_p, blocksize_p >> > (fbuf,output);
	auto input2 = vector<float3>(particleNum);
	/*hipMemcpy(&input2[0], output, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
	for (auto pos : input2) 
		cout << pos.y << " ";
	cout << endl; 
	hipMemcpy(&input2[0], fbuf.vel_update, particleNum * sizeof(float3), hipMemcpyDeviceToHost);
	for (auto ve : input2)
		cout << ve.y << " ";
	cout << endl;*/
	////Safe
}
void stepTime() {
	float3 * input,* output;
	hipGraphicsMapResources(1, &cuda_vbo_resource[0], 0);
	hipGraphicsMapResources(1, &cuda_vbo_resource[1], 0);


	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&input, &num_bytes,
		cuda_vbo_resource[0]);
	hipGraphicsResourceGetMappedPointer((void **)&output, &num_bytes,
		cuda_vbo_resource[1]);
	//advectParticles(input, output);
	//hipDeviceSynchronize();
	CountParticles(input);
	auto input1 = vector<int>(outerGridNum);

	//hipMemcpy(&input1[0], fbuf.grid_particles_num, GridNum * sizeof(int), hipMemcpyDeviceToHost);
	//for (int a : input1)
	//	cout << a << " ";
	//hipDeviceSynchronize();
	IndexSort(input);
	ComputeOtherForce();
	PredictonCorrection(output);
	Advance(output);
	
	
	hipGraphicsUnmapResources(1, &cuda_vbo_resource[0], 0);
	hipGraphicsUnmapResources(1, &cuda_vbo_resource[1], 0);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	swapBuff();
}
uint particleSystem::getParticleNum()
{
	return particleNum;
}
float particleSystem::getRadius() {
	return radius;
}
float particleSystem::getSmoothRadius() {
	return smoothRadius;
}

__global__ void scanSumInt(int * input, int * output, int *aux,int numPerthread) {
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	int offset = id * numPerthread;
	int prefix=0;
	for (int i = offset; i < offset + numPerthread; i++) {
		int tmp = input[i];
		output[i] = prefix;
		prefix += tmp;
	}
	//if(aux)
	//for (int i = offset; i < offset + numPerthread; i++) {
	if(aux)
		aux[id] = prefix;
	
}
__global__ void addUpPrefix(int* prefix, int* aux ,int stride) {
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	prefix[id] += aux[id/stride];
}

void prescanInt(int* input, int* output, int len, int numPerThread,int numBlock,int numThread) {
	int * aux;
	//auto input1 = vector<int>(GridNum);

	int totalthread =  (numBlock*numThread);
	CUDA_SAFE_CALL(hipMalloc(&aux, totalthread *sizeof(int)));
	hipDeviceSynchronize();

	//hipMemcpy(&input1[0], input, GridNum * sizeof(int), hipMemcpyDeviceToHost);

	int sum = 0;
	/*for (int a : input1)
		cout << (sum=sum+a) << " ";
	hipDeviceSynchronize();*/
	scanSumInt << <numBlock, numThread >> > (input, output,aux,numPerThread);
	

	scanSumInt << <1, 1 >> > (aux, aux, NULL, totalthread);  //in place 
	hipDeviceSynchronize();
	
	hipDeviceSynchronize();
	addUpPrefix<<<numBlock*numThread,  numPerThread >>>(output, aux, numPerThread);
	CUDA_SAFE_CALL(hipFree(aux));

	//hipMemcpy(&input1[0], output, GridNum * sizeof(int), hipMemcpyDeviceToHost);
	//
	//
	//for (int a : input1)
	//	cout << a << " ";
	//hipDeviceSynchronize();

	
}

//__global__ void reduceMax(float* input,float*output, int num) {
//	__shared__ float table[num];
//	int total = num;
//	int tid = threadIdx.x;
//	while (total>5)
//	{
//		__syncthreads();
//	}
//
//}

void testf() {
	int data[6000];
	/*for (int i = 0; i < 6000; i++)
		data[i] = i;
	int result = thrust::reduce(data, data + 6000,
		-1,
		thrust::maximum<int>());
	cout << result << endl;
	float input[125];
	for (int i = 0; i < 125; i++)
		input[i] = rand()%125;
	for (int a : input)
		cout << a << " ";
	cout << endl;*/
	//float* dev_a,* dev_b;
	//CUDA_SAFE_CALL(hipMalloc(&dev_a,125*sizeof(float)));
	//CUDA_SAFE_CALL(hipMalloc(&dev_b, sizeof(float)));
	//////hipDeviceSynchronize();

	//CUDA_SAFE_CALL(hipMemcpy(dev_a, input, 125 * sizeof(float), hipMemcpyHostToDevice));
	//ReduceMax(dev_a, dev_b, 125);
	//////prescanInt(dev_a, dev_b, 125, 5, 5, 5);
	////hipDeviceSynchronize();
	//CUDA_SAFE_CALL(hipDeviceSynchronize());
	//float *res=new float[1];
	//////hipMemset(dev_b, 0, sizeof(float));
	//hipMemcpy(res, dev_b,  sizeof(float), hipMemcpyDeviceToHost);
	//cout << res[0] << endl;
}
int getghostNum() {
	return ghostnum;
}

